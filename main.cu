#include <iostream>
#include <hip/hip_runtime.h>

__global__ void addKernel(int *c, const int *a, const int *b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    const int size = 10;
    const int bytes = size * sizeof(int);

    int h_a[size] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
    int h_b[size] = {10, 9, 8, 7, 6, 5, 4, 3, 2, 1};
    int h_c[size];

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    addKernel<<<1, size>>>(d_c, d_a, d_b, size);

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    std::cout << "Result:\n";
    for (int i = 0; i < size; ++i) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}