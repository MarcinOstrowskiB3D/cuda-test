#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addKernel(int *c, const int *a, const int *b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    constexpr int kSize = 10;
    const int bytes = kSize * sizeof(int);

    int h_a[kSize] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
    int h_b[kSize] = {10, 9, 8, 7, 6, 5, 4, 3, 2, 1};
    int h_c[kSize];

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    addKernel<<<1, kSize>>>(d_c, d_a, d_b, kSize);

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    std::cout << "Result:\n";
    for (int i = 0; i < kSize; ++i) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
